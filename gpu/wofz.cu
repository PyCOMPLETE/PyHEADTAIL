/////////////////////////////////////////////////////////////////////////////
//
// DATE
//   06/22/2015
//
// AUTHORS
//   Hannes Bartosik, Adrian Oeftiger
//
// DESCRIPTION
//   FADDEEVA error function for GPU in CUDA.
//   This file is intended to be used as a
//   preamble to depending kernels, e.g. in PyCUDA
//   via ElementwiseKernel(..., preamble=open( <this_file> ).read()).
//
/////////////////////////////////////////////////////////////////////////////


#include <hip/hip_runtime.h>
#include <math.h>

#define errf_const 1.12837916709551
#define xLim 5.33
#define yLim 4.29

__device__ void wofz(double in_real, double in_imag,
                     double* out_real, double* out_imag)
{
    /**
    this function calculates the double precision complex error function
    based on the algorithm of the FORTRAN function written at CERN by
    K. Koelbig, Program C335, 1970.

    See also M. Bassetti and G.A. Erskine, "Closed expression for the
    electric field of a two-dimensional Gaussian charge density",
    CERN-ISR-TH/80-06.
    */

    int n, nc, nu;
    double h, q, Saux, Sx, Sy, Tn, Tx, Ty, Wx, Wy, xh, xl, x, yh, y;
    double Rx [33];
    double Ry [33];

    x = fabs(in_real);
    y = fabs(in_imag);

    if (y < yLim && x < xLim) {
        q = (1.0 - y / yLim) * sqrt(1.0 - (x / xLim) * (x / xLim));
        h  = 1.0 / (3.2 * q);
        nc = 7 + int(23.0 * q);
        xl = pow(h, double(1 - nc));
        xh = y + 0.5 / h;
        yh = x;
        nu = 10 + int(21.0 * q);
        Rx[nu] = 0.;
        Ry[nu] = 0.;
        for (n = nu; n > 0; n--){
            Tx = xh + n * Rx[n];
            Ty = yh - n * Ry[n];
            Tn = Tx*Tx + Ty*Ty;
            Rx[n-1] = 0.5 * Tx / Tn;
            Ry[n-1] = 0.5 * Ty / Tn;
            }
        Sx = 0.;
        Sy = 0.;
        for (n = nc; n>0; n--){
            Saux = Sx + xl;
            Sx = Rx[n-1] * Saux - Ry[n-1] * Sy;
            Sy = Rx[n-1] * Sy + Ry[n-1] * Saux;
            xl = h * xl;
        };
        Wx = errf_const * Sx;
        Wy = errf_const * Sy;
    }
    else {
        xh = y;
        yh = x;
        Rx[0] = 0.;
        Ry[0] = 0.;
        for (n = 9; n>0; n--){
            Tx = xh + n * Rx[0];
            Ty = yh - n * Ry[0];
            Tn = Tx * Tx + Ty * Ty;
            Rx[0] = 0.5 * Tx / Tn;
            Ry[0] = 0.5 * Ty / Tn;
        };
        Wx = errf_const * Rx[0];
        Wy = errf_const * Ry[0];
    }

    if (y == 0.) {
        Wx = exp(-x * x);
    }
    if (in_imag < 0.) {
        Wx =   2.0 * exp(y * y - x * x) * cos(2.0 * x * y) - Wx;
        Wy = - 2.0 * exp(y * y - x * x) * sin(2.0 * x * y) - Wy;
        if (in_real > 0.) {
            Wy = -Wy;
        }
    }
    else if (in_real < 0.) {
        Wy = -Wy;
    }

    *out_real = Wx;
    *out_imag = Wy;
}
